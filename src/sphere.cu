#include "hip/hip_runtime.h"
#include "sphere.h"

auto timeMillisecond()
{
    using namespace chrono;
    return duration_cast<milliseconds>(system_clock::now().time_since_epoch()).count();
}

inline float sumSquare(float a, float b, float c)
{
    float re = (a * a) + (b * b) + (c * c);
    return re;
}

inline string f2s(float x, int precision = 3) 
{
    return number2string<float>(x, precision);
}

template <typename T> inline string number2string(T x, int precision)
{
    stringstream re;
    re << fixed << setprecision(precision) << setw(8) << x;
    return re.str();
}

inline vector<string> vF2vS(vector<float> x, int precision = 3) 
{
    return vector2vecString<float>(x, precision);
}

template <typename T> inline vector<string> vector2vecString(vector<T> x, int precision)
{
    vector<string> re;
    for (int i=0; i<x.size(); i++) {
        re.push_back(number2string<T>(x[i],precision));
    }
    return re;
}

set<vector<string>> loopSphere(float radius, float resolution, bool debug = false)
{
    RECURSION_SET.clear();
    auto timeStart = timeMillisecond();
    set<vector<string>> re;
    if (radius <= 0) {
        return re;
    }

    if (resolution > radius) {
        resolution = radius;
    }

    float limitSquareMax = radius + resolution;
    float limitSquareMin = radius - resolution;

    int len = int(radius / resolution);
    for (int i = 0; i < len+1; i++) {
        float i_z = i * resolution;
        float i_x = len * resolution;
        while (i_x >= i_z) {
            float i_y = 0.0;
            while (i_x >= i_y) {
                float s = sumSquare(i_x, i_y, i_z);
                if ((s >= limitSquareMin) && (s <= limitSquareMax)) {
                    RECURSION_SET.insert( vector<string> {f2s(i_x), f2s(i_y), f2s(i_z)});
                    RECURSION_SET.insert( vector<string> {f2s(i_y), f2s(i_x), f2s(i_z)});
                    RECURSION_SET.insert( vector<string> {f2s(i_z), f2s(i_y), f2s(i_x)});
                }
                i_y += resolution;
            }
            i_x -= resolution;
        }
    }

    if (!debug) {
        for (vector<vector<int>>::iterator s = OCTANTS.begin(); 
             s != OCTANTS.end(); s++) 
        {
            for (set<vector<string>>::iterator p = RECURSION_SET.begin(); 
                 p!= RECURSION_SET.end(); p++) 
            {
                vector<string> vec = {  f2s( (*s)[0] * stof( (*p)[0] ) ), 
                                        f2s( (*s)[1] * stof( (*p)[1] ) ), 
                                        f2s( (*s)[2] * stof( (*p)[2] ) )};
                re.insert(vec);
            }
        }
    } else {
        re = RECURSION_SET;
    }

    cout << re.size() << endl; 
    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Duration (ms) [function: loopSphere]: " << timeDuration << endl; 
    return re;
}

set<vector<string>> recursionSphere(float radius, float resolution, bool debug = false)
{
    RECURSION_SET.clear();
    auto timeStart = timeMillisecond();
    set<vector<string>> re;

    float limitSquareMax = radius + resolution;
    float limitSquareMin = radius - resolution;

    vector<float> p0 = {radius, 0.0, 0.0};
    recursionSphere(p0, radius, resolution, limitSquareMin, limitSquareMax);
    
    if (!debug) {
        for (vector<vector<int>>::iterator s = OCTANTS.begin(); 
             s != OCTANTS.end(); s++) 
        {
            for (set<vector<string>>::iterator p = RECURSION_SET.begin(); 
                 p!= RECURSION_SET.end(); p++) 
            {
                vector<string> vec = {  f2s( (*s)[0] * stof( (*p)[0] ) ), 
                                        f2s( (*s)[1] * stof( (*p)[1] ) ), 
                                        f2s( (*s)[2] * stof( (*p)[2] ) )};
                re.insert(vec);
            }
        }
    } else {
        re = RECURSION_SET;
    }

    cout << re.size() << endl; 
    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Duration (ms) [function: recursionSphere]: " << timeDuration << endl; 
    return re;
}

void recursionSphere(vector<float> p, float r, float resolution, float min, float max)
{
    if (p[0]<0 || p[1]<0 || p[2]<0 || p[0]>r || p[1]>r || p[2]>r) {
        return;
    }
    float s = sumSquare(p[0], p[1], p[2]);
    if ((s >= min) && (s <= max)) {
        RECURSION_SET.insert(vF2vS(p));
    }

    for (int i = 0; i < 3; i++) {
        vector<float> p1 = p;
        p1[i] = p[i] + resolution;
        if ((s <= max) && !(RECURSION_SET.find(vF2vS(p1)) != RECURSION_SET.end())) {
            recursionSphere(p1, r, resolution, min, max);
        }
        p1 = p;
        p1[i] = p[i] - resolution;
        if ((s >= min) && !(RECURSION_SET.find(vF2vS(p1)) != RECURSION_SET.end())) {
            recursionSphere(p1, r, resolution, min, max);
        }
    }

    return;
}

__global__ 
void cudaInitArr(int* arr, 
                 int ROW, 
                 int len, 
                 float resolution, 
                 float max, float min)
{
    // int i = threadIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < ROW; i+=stride) {
        // 1. grows in z direction, from 0 to len
        float z = float( i % (len+1))                   * resolution;
        // 2. grows in y direction, from 0 to len
        float y = float( i/(len+1) % (len+1))           * resolution;
        // 3. grows in x direction, from 0 to len
        float x = float( i/((len+1)*(len+1)) % (len+1)) * resolution;
        // float y = float( (i%((len+1)*(len+1))) / (len+1)) * resolution;
        
        float sum = (x*x)+(y*y)+(z*z);
        if (sum>=min && sum<=max) {
            arr[i] = 1;
        } else {
            arr[i] = 0;
        }
    }
}

void cpuInitArr(int* arr, int ROW, int len, float resolution, float max, float min)
{
    // int i = threadIdx.x;
    for (int i = 0; i < ROW; i++) {
        // 1. grows in z direction, from 0 to len
        float z = float( i % (len+1))                   * resolution;
        // 2. grows in y direction, from 0 to len
        float y = float( i/(len+1) % (len+1))           * resolution;
        // 3. grows in x direction, from 0 to len
        float x = float( i/((len+1)*(len+1)) % (len+1)) * resolution;
        // float y = float( (i%((len+1)*(len+1))) / (len+1)) * resolution;
        
        float sum = (x*x)+(y*y)+(z*z);
        if (sum>=min && sum<=max) {
            arr[i] = 1;
        } else {
            arr[i] = 0;
        }
    }
}

set<vector<string>> loopSphereBruteForceCUDA(float radius, float resolution, bool debug = false)
{
    RECURSION_SET.clear();
    auto timeStart = timeMillisecond();
    set<vector<string>> re;
    if (radius <= 0) {
        return re;
    }

    if (resolution > radius) {
        resolution = radius;
    }

    float limitSquareMax = radius + resolution;
    float limitSquareMin = radius - resolution;

    int len = int(radius / resolution);
    int ROW = (len + 1) * (len + 1) * (len + 1); 
    int   *intArr,   *gpu_intArr;

    
    #define BLOCK_SIZE 256
    #define GRID_SIZE 100

    intArr = (int*)malloc(sizeof(int) * ROW);
    hipMalloc((void**)&gpu_intArr, sizeof(int) * ROW);
    // hipMemcpy(gpu_intArr, intArr, sizeof(int) * ROW, hipMemcpyHostToDevice);

    cudaInitArr<<<GRID_SIZE, BLOCK_SIZE>>>
        (gpu_intArr, ROW, len, resolution, limitSquareMax, limitSquareMin);
    hipMemcpy(intArr, gpu_intArr, sizeof(int) * ROW, hipMemcpyDeviceToHost);

    for (int i = 0; i < ROW; i++) {
        if (intArr[i] == 1) {
            float z = resolution * (i % (len + 1));
            float y = resolution * (i/(len+1) %(len+1));
            float x = resolution * (i/((len+1)*(len+1)) % (len+1));
            RECURSION_SET.insert( vector<string> {f2s(x), f2s(y), f2s(z)});
        } 
    }

    if (!debug) {
        for (vector<vector<int>>::iterator s = OCTANTS.begin(); 
             s != OCTANTS.end(); s++) 
        {
            for (set<vector<string>>::iterator p = RECURSION_SET.begin(); 
                 p!= RECURSION_SET.end(); p++) 
            {
                vector<string> vec = {  f2s( (*s)[0] * stof( (*p)[0] ) ), 
                                        f2s( (*s)[1] * stof( (*p)[1] ) ), 
                                        f2s( (*s)[2] * stof( (*p)[2] ) )};
                re.insert(vec);
            }
        }
    } else {
        re = RECURSION_SET;
    }

    cout << re.size() << endl; 
    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Duration (ms) [function: loopSphereBruteForceCUDA]: " << timeDuration << endl; 
    return re;
}

set<vector<string>> loopSphereBruteForce(float radius, float resolution, bool debug = false)
{
    RECURSION_SET.clear();
    auto timeStart = timeMillisecond();
    set<vector<string>> re;
    if (radius <= 0) {
        return re;
    }

    if (resolution > radius) {
        resolution = radius;
    }

    float limitSquareMax = radius + resolution;
    float limitSquareMin = radius - resolution;

    int len = int(radius / resolution);
    int ROW = (len + 1) * (len + 1) * (len + 1); 
    int   *intArr;


    intArr = (int*)malloc(sizeof(int) * ROW);

    cpuInitArr(intArr, ROW, len, resolution, limitSquareMax, limitSquareMin);

    for (int i = 0; i < ROW; i++) {
        if (intArr[i] == 1) {
            float z = resolution * (i % (len + 1));
            float y = resolution * (i/(len+1) %(len+1));
            float x = resolution * (i/((len+1)*(len+1)) % (len+1));
            RECURSION_SET.insert( vector<string> {f2s(x), f2s(y), f2s(z)});
        } 
    }

    if (!debug) {
        for (vector<vector<int>>::iterator s = OCTANTS.begin(); 
             s != OCTANTS.end(); s++) 
        {
            for (set<vector<string>>::iterator p = RECURSION_SET.begin(); 
                 p!= RECURSION_SET.end(); p++) 
            {
                vector<string> vec = {  f2s( (*s)[0] * stof( (*p)[0] ) ), 
                                        f2s( (*s)[1] * stof( (*p)[1] ) ), 
                                        f2s( (*s)[2] * stof( (*p)[2] ) )};
                re.insert(vec);
            }
        }
    } else {
        re = RECURSION_SET;
    }

    cout << re.size() << endl; 
    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Duration (ms) [function: loopSphereBruteForce]: " << timeDuration << endl; 
    return re;    
}

int writeToPDB(set<vector<string>> totPoints, string fname = "sphere_test_out.pdb")
{
    ofstream fo;
    fo.open(fname);
    fo << "REMARK this is a sphere\n";
    int i_ctrl = 0;
    for (set<vector<string>>::iterator p = totPoints.begin(); 
         p!= totPoints.end(); p++) 
    {
        int resid = i_ctrl+1;
        if (resid > 9999) {
            resid = resid % 10000;
        }
        string chain = "A";
        if ((*p).size() < 4) {
            chain = "A";
        } else {
            chain = (*p)[3];
        }

        fo << "ATOM " 
           << setw(6) <<  i_ctrl+1 
           << " C    DUM " << chain 
           << setw(4) << resid 
           << "    " 
           << (*p)[0]
           << (*p)[1] 
           << (*p)[2]
           << "  1.00  0.00  " << endl;
        i_ctrl += 1;
    }
    fo << "TER\n" ;
    fo.close();
    return 0;
}

void benchmarkTest()
{
    cout << "====Benchmark====" << endl;
    // Time(ms): 682
    // Points: 127560
    set<vector<string>> totPoints = loopSphere(1.0, 0.01);
    writeToPDB(totPoints, "benchmark_loop_DnC.pdb");

    // Time(ms): 14581
    // Points: 127752
    // totPoints = recursionSphere(1.0, 0.01);
    // writeToPDB(totPoints, "benchmark_recursion.pdb");

    // Time(ms): 490
    // Points: 127840
    totPoints = loopSphereBruteForceCUDA(1.0, 0.01);
    writeToPDB(totPoints, "benchmark_loop_BF_CUDA.pdb");

    // Time(ms): 506
    // Points: 127856
    totPoints = loopSphereBruteForce(1.0, 0.01);
    writeToPDB(totPoints, "benchmark_loop_BF.pdb");

    cout << "====Benchmark Finished====\n" << endl;
}

void debugTest()
{
    cout << "====Debug====" << endl;   
    set<vector<string>> totPoints;

    totPoints = loopSphereBruteForceCUDA(1.0, 0.01, true);
    writeToPDB(totPoints, "debug_loop_BF_CUDA.pdb");

    totPoints = loopSphereBruteForce(1.0, 0.01, true);
    writeToPDB(totPoints, "debug_loop_BF.pdb");
    
    totPoints = loopSphere(1.0, 0.01, true);
    writeToPDB(totPoints, "debug_loop.pdb");

    // totPoints = recursionSphere(1.0, 0.01, true);
    // writeToPDB(totPoints, "debug_recursion.pdb");
    cout << "====Debug Finished====\n" << endl;
}

int main(int argc, char **argv)
{
    auto timeStart = timeMillisecond();

    debugTest();
    benchmarkTest();

    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Runtime (ms): " << timeDuration << endl; 
    return 0;
}