#include "hip/hip_runtime.h"
#include "sphereLoopBFCUDA.h"

using namespace std;

__global__ 
void cudaInitArr(int* arr, 
                 int ROW, 
                 int len, 
                 float resolution, 
                 float max, float min)
{
    // int i = threadIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < ROW; i+=stride) {
        // 1. grows in z direction, from 0 to len
        float z = float( i % (len+1))                   * resolution;
        // 2. grows in y direction, from 0 to len
        float y = float( i/(len+1) % (len+1))           * resolution;
        // 3. grows in x direction, from 0 to len
        float x = float( i/((len+1)*(len+1)) % (len+1)) * resolution;
        // float y = float( (i%((len+1)*(len+1))) / (len+1)) * resolution;
        
        float sum = (x*x)+(y*y)+(z*z);
        if (sum>=min && sum<=max) {
            arr[i] = 1;
        } else {
            arr[i] = 0;
        }
    }
}

set<vector<string>> loopSphereBruteForceCUDA(float radius, float resolution)
{
    return loopSphereBruteForceCUDA(radius, resolution, false);
}


set<vector<string>> loopSphereBruteForceCUDA(float radius, float resolution, bool debug = false)
{
    std::vector<std::vector<int>> OCTANTS = {
        {1,1,+1}, {-1,1,+1}, {-1,-1,+1}, {1,-1,+1},
        {1,1,-1}, {-1,1,-1}, {-1,-1,-1}, {1,-1,-1}
    };
    std::set<std::vector<std::string>> RECURSION_SET;

    RECURSION_SET.clear();
    auto timeStart = timeMillisecond();
    set<vector<string>> re;
    if (radius <= 0) {
        return re;
    }

    if (resolution > radius) {
        resolution = radius;
    }

    float limitSquareMax = radius + resolution;
    float limitSquareMin = radius - resolution;

    int len = int(radius / resolution);
    int ROW = (len + 1) * (len + 1) * (len + 1); 
    int   *intArr,   *gpu_intArr;

    
    #define BLOCK_SIZE 256
    #define GRID_SIZE 100

    intArr = (int*)malloc(sizeof(int) * ROW);
    hipMalloc((void**)&gpu_intArr, sizeof(int) * ROW);
    // hipMemcpy(gpu_intArr, intArr, sizeof(int) * ROW, hipMemcpyHostToDevice);

    cudaInitArr<<<GRID_SIZE, BLOCK_SIZE>>>
        (gpu_intArr, ROW, len, resolution, limitSquareMax, limitSquareMin);
    hipMemcpy(intArr, gpu_intArr, sizeof(int) * ROW, hipMemcpyDeviceToHost);

    for (int i = 0; i < ROW; i++) {
        if (intArr[i] == 1) {
            float z = resolution * (i % (len + 1));
            float y = resolution * (i/(len+1) %(len+1));
            float x = resolution * (i/((len+1)*(len+1)) % (len+1));
            RECURSION_SET.insert( vector<string> {f2s(x), f2s(y), f2s(z)});
        } 
    }

    if (!debug) {
        for (vector<vector<int>>::iterator s = OCTANTS.begin(); 
             s != OCTANTS.end(); s++) 
        {
            for (set<vector<string>>::iterator p = RECURSION_SET.begin(); 
                 p!= RECURSION_SET.end(); p++) 
            {
                vector<string> vec = {  f2s( (*s)[0] * stof( (*p)[0] ) ), 
                                        f2s( (*s)[1] * stof( (*p)[1] ) ), 
                                        f2s( (*s)[2] * stof( (*p)[2] ) )};
                re.insert(vec);
            }
        }
    } else {
        re = RECURSION_SET;
    }

    cout << re.size() << endl; 
    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Duration (ms) [function: loopSphereBruteForceCUDA]: " << timeDuration << endl; 
    return re;
}

