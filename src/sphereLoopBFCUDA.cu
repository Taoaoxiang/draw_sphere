#include "hip/hip_runtime.h"
#include "sphereLoopBFCUDA.h"

using namespace std;

__global__ 
void cudaInitArr(int* arr, 
                 int ROW, 
                 int len, 
                 float resolution, 
                 float max, float min)
{
    // int i = threadIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < ROW; i+=stride) {
        // 1. grows in z direction, from 0 to len
        float z = float( i % (len+1))                   * resolution;
        // 2. grows in y direction, from 0 to len
        float y = float( i/(len+1) % (len+1))           * resolution;
        // 3. grows in x direction, from 0 to len
        float x = float( i/((len+1)*(len+1)) % (len+1)) * resolution;
        // float y = float( (i%((len+1)*(len+1))) / (len+1)) * resolution;
        
        float sum = (x*x)+(y*y)+(z*z);
        if (sum>=min && sum<=max) {
            arr[i] = 1;
        } else {
            arr[i] = 0;
        }
    }
}

__global__ 
void cudaInitArr(int* arr, 
                 float *coor,
                 int ROW, 
                 int len, 
                 float resolution, 
                 float max, float min)
{
    // int i = threadIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < ROW; i+=stride) {
        // 1. grows in z direction, from 0 to len
        float z = float( i % (len+1))                   * resolution;
        // 2. grows in y direction, from 0 to len
        float y = float( i/(len+1) % (len+1))           * resolution;
        // 3. grows in x direction, from 0 to len
        float x = float( i/((len+1)*(len+1)) % (len+1)) * resolution;
        // float y = float( (i%((len+1)*(len+1))) / (len+1)) * resolution;
        
        float sum = (x*x)+(y*y)+(z*z);
        if (sum>=min && sum<=max) {
            arr[i] = 1;
            coor[i] = x;
            coor[i+ROW] = y;
            coor[i+ROW+ROW] = z;
        } else {
            arr[i] = 0;
        }
    }
}

__global__ 
void cudaInitArrAll(int* arr, 
                 float *coor,
                 int ROW, 
                 int len, 
                 float resolution, 
                 float max, float min)
{
    // int i = threadIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < ROW; i+=stride) {
        // 1. grows in z direction, from 0 to len
        float z = (float( i % (2*len+1)) - len)                 * resolution;
        // 2. grows in y direction, from 0 to len
        float y = (float( i/(2*len+1) % (2*len+1))-len)           * resolution;
        // 3. grows in x direction, from 0 to len
        float x = (float( i/((2*len+1)*(2*len+1)) % (2*len+1))-len) * resolution;
        
        float sum = (x*x)+(y*y)+(z*z);
        if (sum>=min && sum<=max) {
            arr[i] = 1;
            coor[i] = x;
            coor[i+ROW] = y;
            coor[i+ROW+ROW] = z;
        } else {
            arr[i] = 0;
        }
    }
}

set<vector<string>> loopSphereBruteForceCUDA(float radius, float resolution)
{
    return loopSphereBruteForceCUDA(radius, resolution, false);
}


set<vector<string>> loopSphereBruteForceCUDA(float radius, float resolution, bool debug = false)
{
    std::vector<std::vector<int>> OCTANTS = {
        {1,1,+1}, {-1,1,+1}, {-1,-1,+1}, {1,-1,+1},
        {1,1,-1}, {-1,1,-1}, {-1,-1,-1}, {1,-1,-1}
    };
    std::set<std::vector<std::string>> RECURSION_SET;

    RECURSION_SET.clear();
    auto timeStart = timeMillisecond();
    set<vector<string>> re;
    if (radius <= 0) {
        return re;
    }

    if (resolution > radius) {
        resolution = radius;
    }

    float limitSquareMax = radius + resolution;
    float limitSquareMin = radius - resolution;

    int len = int(radius / resolution);
    int ROW = (len + 1) * (len + 1) * (len + 1); 
    int   *intArr,   *gpu_intArr;

    
    #define BLOCK_SIZE 256
    #define GRID_SIZE 100

    intArr = (int*)malloc(sizeof(int) * ROW);
    hipMalloc((void**)&gpu_intArr, sizeof(int) * ROW);
    // hipMemcpy(gpu_intArr, intArr, sizeof(int) * ROW, hipMemcpyHostToDevice);

    cudaInitArr<<<GRID_SIZE, BLOCK_SIZE>>>
        (gpu_intArr, ROW, len, resolution, limitSquareMax, limitSquareMin);
    hipMemcpy(intArr, gpu_intArr, sizeof(int) * ROW, hipMemcpyDeviceToHost);

    hipFree(gpu_intArr);

    for (int i = 0; i < ROW; i++) {
        if (intArr[i] == 1) {
            float z = resolution * (i % (len + 1));
            float y = resolution * (i/(len+1) %(len+1));
            float x = resolution * (i/((len+1)*(len+1)) % (len+1));
            RECURSION_SET.insert( vector<string> {f2s(x), f2s(y), f2s(z)});
        } 
    }

    if (!debug) {
        for (vector<vector<int>>::iterator s = OCTANTS.begin(); 
             s != OCTANTS.end(); s++) 
        {
            for (set<vector<string>>::iterator p = RECURSION_SET.begin(); 
                 p!= RECURSION_SET.end(); p++) 
            {
                vector<string> vec = {  f2s( (*s)[0] * stof( (*p)[0] ) ), 
                                        f2s( (*s)[1] * stof( (*p)[1] ) ), 
                                        f2s( (*s)[2] * stof( (*p)[2] ) )};
                re.insert(vec);
            }
        }
    } else {
        re = RECURSION_SET;
    }

    cout << re.size() << endl; 
    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Duration (ms) [function: loopSphereBruteForceCUDA]: " << timeDuration << endl; 
    return re;
}

set<vector<string>> loopSphereBruteForce2CUDA(float radius, float resolution)
{
    return loopSphereBruteForce2CUDA(radius, resolution, false);
}

set<vector<string>> loopSphereBruteForce2CUDA(float radius, float resolution, bool debug = false)
{
    std::vector<std::vector<int>> OCTANTS = {
        {1,1,+1}, {-1,1,+1}, {-1,-1,+1}, {1,-1,+1},
        {1,1,-1}, {-1,1,-1}, {-1,-1,-1}, {1,-1,-1}
    };
    std::set<std::vector<std::string>> RECURSION_SET;

    RECURSION_SET.clear();
    auto timeStart = timeMillisecond();
    set<vector<string>> re;
    if (radius <= 0) {
        return re;
    }

    if (resolution > radius) {
        resolution = radius;
    }

    float limitSquareMax = radius + resolution;
    float limitSquareMin = radius - resolution;

    int len = int(radius / resolution);
    int ROW = (len + 1) * (len + 1) * (len + 1); 
    int   *intArr,   *gpu_intArr;
    float *coor, *gpu_coor;
    
    #define BLOCK_SIZE 256
    #define GRID_SIZE 100

    intArr = (int*)malloc(sizeof(int) * ROW);
    hipMalloc((void**)&gpu_intArr, sizeof(int) * ROW);
    // hipMemcpy(gpu_intArr, intArr, sizeof(int) * ROW, hipMemcpyHostToDevice);

    // coor: 2D array (N,3) to 1D array (N*3)
    coor = (float*)malloc(sizeof(float*) * ROW * 3);
    hipMalloc((void**)&gpu_coor, sizeof(float) * ROW * 3);

    cudaInitArr<<<GRID_SIZE, BLOCK_SIZE>>>
        (gpu_intArr, gpu_coor, ROW, len, resolution, limitSquareMax, limitSquareMin);

    hipMemcpy(intArr, gpu_intArr, sizeof(int) * ROW, hipMemcpyDeviceToHost);
    hipMemcpy(coor, gpu_coor, sizeof(float) * ROW * 3,hipMemcpyDeviceToHost); 
   
    hipFree(gpu_intArr);
    hipFree(gpu_coor);

    for (int i = 0; i < ROW; i++) {
        if (intArr[i] == 1) {
            float x = coor[i];
            float y = coor[i+ROW];
            float z = coor[i+ROW+ROW];
            RECURSION_SET.insert( vector<string> {f2s(x), f2s(y), f2s(z)});
        } 
    }
    re = RECURSION_SET;

    if (!debug) {
        for (vector<vector<int>>::iterator s = OCTANTS.begin(); 
             s != OCTANTS.end(); s++) 
        {
            for (set<vector<string>>::iterator p = RECURSION_SET.begin(); 
                 p!= RECURSION_SET.end(); p++) 
            {
                vector<string> vec = {  f2s( (*s)[0] * stof( (*p)[0] ) ), 
                                        f2s( (*s)[1] * stof( (*p)[1] ) ), 
                                        f2s( (*s)[2] * stof( (*p)[2] ) )};
                re.insert(vec);
            }
        }
    } else {
        re = RECURSION_SET;
    }

    cout << re.size() << endl; 
    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Duration (ms) [function: loopSphereBruteForce2CUDA]: " << timeDuration << endl; 
    return re;
}

set<vector<string>> loopSphereBruteForceAllCUDA(float radius, float resolution)
{
    return loopSphereBruteForceAllCUDA(radius, resolution, false);
}

set<vector<string>> loopSphereBruteForceAllCUDA(float radius, float resolution, bool debug = false)
{

    std::set<std::vector<std::string>> RECURSION_SET;

    RECURSION_SET.clear();
    auto timeStart = timeMillisecond();
    set<vector<string>> re;
    if (radius <= 0) {
        return re;
    }

    if (resolution > radius) {
        resolution = radius;
    }

    float limitSquareMax = radius + resolution;
    float limitSquareMin = radius - resolution;

    int len = int(radius / resolution);
    int ROW = (2*len + 1) * (2*len + 1) * (2*len + 1); 
    int   *intArr,   *gpu_intArr;
    float *coor, *gpu_coor;
    
    #define BLOCK_SIZE 256
    #define GRID_SIZE 100

    intArr = (int*)malloc(sizeof(int) * ROW);
    hipMalloc((void**)&gpu_intArr, sizeof(int) * ROW);
    // hipMemcpy(gpu_intArr, intArr, sizeof(int) * ROW, hipMemcpyHostToDevice);

    // coor: 2D array (N,3) to 1D array (N*3)
    coor = (float*)malloc(sizeof(float*) * ROW * 3);
    hipMalloc((void**)&gpu_coor, sizeof(float) * ROW * 3);

    cudaInitArrAll<<<GRID_SIZE, BLOCK_SIZE>>>
        (gpu_intArr, gpu_coor, ROW, len, resolution, limitSquareMax, limitSquareMin);

    hipMemcpy(intArr, gpu_intArr, sizeof(int) * ROW, hipMemcpyDeviceToHost);
    hipMemcpy(coor, gpu_coor, sizeof(float) * ROW * 3,hipMemcpyDeviceToHost); 
   
    for (int i = 0; i < ROW; i++) {
        if (intArr[i] == 1) {
            float x = coor[i];
            float y = coor[i+ROW];
            float z = coor[i+ROW+ROW];
            RECURSION_SET.insert( vector<string> {f2s(x), f2s(y), f2s(z)});
        } 
    }
    re = RECURSION_SET;

    hipFree(gpu_intArr);
    hipFree(gpu_coor);

    cout << re.size() << endl; 
    auto timeDuration = timeMillisecond() - timeStart;
    cout << "Duration (ms) [function: loopSphereBruteForceAllCUDA]: " << timeDuration << endl; 
    return re;
}



